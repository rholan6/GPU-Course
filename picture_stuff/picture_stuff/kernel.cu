#include "hip/hip_runtime.h"
//since threshold has a gpu and cpu version, may as well have a way to not setup cuda if we only want the cpu version
#define GPU_VER

#ifdef GPU_VER
#include "hip/hip_runtime.h"
#include ""
#endif

#include <stdio.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

using namespace cv;

__global__ void thresholdKernel(unsigned char* original, unsigned char* modified, unsigned char threshold)
{
	int current = blockIdx.x * gridDim.x + threadIdx.x;
	if (original[current] > threshold) {
		modified[current] = 255;
	}
	else {
		modified[current] = 0;
	}
}

void threshold(unsigned char threshold, Mat& image);
hipError_t GPUThreshold(unsigned char threshold, Mat& original);

int main(int argc, char* argv[])
{
	if (argc != 2) {
		printf("Usage: %s ImageToLoadAndDisplay\n", argv[0]);
		exit(1);
	}

	Mat image;
	image = imread(argv[1], CV_LOAD_IMAGE_COLOR);

	printf("Number of channels: %d\n", image.channels());

	if (!image.data) {
		printf("Could not find or open the image\n");
		exit(1);
	}

	cvtColor(image, image, COLOR_RGB2GRAY);

	unsigned char THRESHOLD = 100;

#ifdef GPU_VER
	hipError_t cudaStatus;

	cudaStatus = GPUThreshold(THRESHOLD, image);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "FAILED to apply threshold filter\n");
		//not sure what state the image will be in after things fail, but it's probably better to just stop
		exit(1);
	}
#endif
#ifndef GPU_VER
	threshold(THRESHOLD, image);
#endif

	namedWindow("Display window", WINDOW_NORMAL);
	imshow("Display window", image);

	waitKey(0);

#ifdef GPU_VER
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "FAILED on hipDeviceReset\n");
		exit(1);
	}
#endif
	return 0;
}

void threshold(unsigned char threshold, Mat& image)
{
	unsigned char* end_data = image.data + (image.rows * image.cols);
	for (unsigned char* p = image.data; p < end_data; p++) {
		if (*p > threshold) {
			*p = 255;
		}
		else {
			*p = 0;
		}
	}
}

hipError_t GPUThreshold(unsigned char threshold, Mat& original)
{
	unsigned char* dev_original = 0;
	unsigned char* dev_modified = 0;
	int* dev_threshold = 0;
	hipError_t cudaStatus;
	int dataSize = original.rows * original.cols * sizeof(unsigned char);
	try
	{
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			throw "FAILED to set CUDA device";
		}

		cudaStatus = hipMalloc((void**)&dev_original, dataSize);
		if (cudaStatus != hipSuccess) {
			throw "FAILED to allocate dev_original";
		}

		cudaStatus = hipMalloc((void**)&dev_modified, dataSize);
		if (cudaStatus != hipSuccess) {
			throw "FAILED to allocate dev_modified";
		}

		cudaStatus = hipMalloc((void**)&dev_threshold, sizeof(int));
		if (cudaStatus != hipSuccess) {
			throw "FAILED to allocate dev_threshold";
		}

		cudaStatus = hipMemcpy(dev_original, original.data, dataSize, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			throw "FAILED to copy image data to GPU memory";
		}

		int numBlocks = original.rows * original.cols / 1024;
		thresholdKernel<<<numBlocks, 1024>>>(dev_original, dev_modified, threshold);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "FAILED launching thresholdKernel: %s\n", hipGetErrorString(cudaStatus));
			throw " ";
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "ERROR after launching thresholdKernel: %d\n", cudaStatus);
			throw " ";
		}
	}
	catch (char* e) {
		fprintf(stderr, "%s\n", e);
	}

	//free memory (might move into catch block)
	if (dev_original != 0) {
		hipFree(dev_original);
	}
	if (dev_modified != 0) {
		hipFree(dev_modified);
	}

	return cudaStatus;
}