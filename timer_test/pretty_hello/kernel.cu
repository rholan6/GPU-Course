#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

hipError_t setup(int* dev_a, int* dev_b, int* dev_c, size_t size);
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
	//exit code (set to 1 in cases of error)
	int retVal = 0;

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
		//print errors to stderr instead of stdout (good practice for larger programs with less important output going to stdout which the user can pipe elsewhere)
		cerr << "addWithCuda failed!" << endl;
		//after printing the error, exit with code 1 (indicating something went wrong)
        retVal = 1;
		goto end_label;
    }

	//leaving this as a printf because the equivalent cout would look terrible
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
		cerr << "hipDeviceReset failed!" << endl;
		//exit with code 1 (to indicate things went south)
        retVal = 1;
		goto end_label;
    }

end_label:
    return retVal;
}

//set up the arrays and CUDA environment
hipError_t setup(int* dev_a, int* dev_b, int* dev_c, size_t size) 
{
	hipError_t cudaStatus;
	try
	{
		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			//cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?" << endl;
			throw "hipSetDevice failed";
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc(&dev_c, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			//cerr << "hipMalloc failed on dev_c!" << endl;
			throw "hipMalloc failed on dev_c";
		}

		cudaStatus = hipMalloc(&dev_a, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			//cerr << "hipMalloc failed on dev_a!" << endl;
			throw "hipMalloc failed on dev_a";
		}

		cudaStatus = hipMalloc(&dev_b, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			//cerr << "hipMalloc failed on dev_b!" << endl;
			throw "hipMalloc failed on dev_b";
		}
	}
	catch (char* e) 
	{
		//catch the exception in more than name alone
		cerr << "Error: " << e << endl;
		//if we run into any problems, free anything that was successfully malloc'd
		if (dev_c != 0) {
			hipFree(dev_c);
		}
		if (dev_a != 0) {
			hipFree(dev_a);
		}
		if (dev_b != 0) {
			hipFree(dev_b);
		}
	}

	//return hipSuccess or whatever error we encountered
	return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

	try
	{
		cudaStatus = setup(dev_a, dev_b, dev_c, size);


		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			cerr << "hipMemcpy failed on a!" << endl;
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			cerr << "hipMemcpy failed on b!" << endl;
			goto Error;
		}

		// Launch a kernel on the GPU with one thread for each element.
		addKernel << <1, size >> > (dev_c, dev_a, dev_b);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			cerr << "addKernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching addKernel!" << endl;
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			cerr << "hipMemcpy failed on dev_c!" << endl;
			goto Error;
		}
	}
	catch (char* e)
	{
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
	}

    return cudaStatus;
}
